#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdlib>
#include <time.h>

#define BLOCK_SIZE 2
#define WA 8
#define HA 8
#define HC 3
#define WC 3
#define WB (WA - WC + 1)
#define HB (HA - HC + 1)


__global__ void Convolution_GPU(float* A, float* B, float* C, int numARows, int numACols, int numBRows, int numBCols, int numCRows, int numCCols)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	// GPU Convolution code
	float sum = 0.0;
	if(row < numBRows && col < numBCols){
		for(int i=0; i < numCRows; i++){
			for (int j=0; j< numCCols; j++){
				sum += A[(row+i) * numARows + (col+j) ] * C[i * numCRows + j];
			}
		}
		B[row * numBRows + col] = sum;
	}
	//

}

void Convolution_CPU(float* A, float* B, float* C, int numARows, int numACols, int numBRows, int numBCols, int numCRows, int numCCols)
{
	// CPU Convolution code
	for(int i=0; i<numBRows; i++){
		for(int j=0; j<numBCols; j++){
			//one cell
			//now one cell calculate
			float tmp = 0.0;
			for(int k_i = 0; k_i < numCRows; k_i++){
				for(int k_j = 0; k_j < numCCols; k_j++){
					// kernel conv
					tmp += A[(i+k_i) * numARows + (j + k_j)] * C[k_i * numCRows + k_j];
				}
			}
			B[i * numBRows + j] = tmp;
		}
	}
	printf("\n");printf("\n");printf("\n");
	//////////////////////////////
}


void randomInit(float* data, int size)
{
	// random initialization code
	for (int i=0; i<size; ++i){
		for (int j=0; j< size; ++j) {
			data[i * size + j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}
	}
	//////////////////////////////
}


int main(int argc, char** argv)
{
	srand(2006);
	hipError_t error;
	hipEvent_t start_G, stop_G, start_C, stop_C;

	hipEventCreate(&start_G);
	hipEventCreate(&stop_G);
	hipEventCreate(&start_C);
	hipEventCreate(&stop_C);

	unsigned int size_A = WA * HA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);

	unsigned int size_B = WB * HB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);

	unsigned int size_C = WC * HC;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C = (float*)malloc(mem_size_C);

	// random initialization
	randomInit(h_A, HA);
	randomInit(h_C, HC);
	//////////////////////////////


	// cudaMalloc
	float *d_A, *d_B, *d_C;
	hipMalloc((void**) &d_A, mem_size_A);
	hipMalloc((void**) &d_B, mem_size_B);
	hipMalloc((void**) &d_C, mem_size_C);

	//////////////////////////////

	// cudaMemcpy
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, mem_size_C, hipMemcpyHostToDevice);
	//////////////////////////////

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((WA) / (BLOCK_SIZE), (WA) / (BLOCK_SIZE));

	hipEventRecord(start_G);

	// GPU Convolution function call
	Convolution_GPU<<<grid, threads>>>(d_A, d_B, d_C, HA, WA, HB, WB, HC, WC);

	//////////////////////////////
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in launching kernel\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	error = hipDeviceSynchronize();

	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipDeviceSynchronize \n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	hipEventRecord(stop_G);
	hipEventSynchronize(stop_G);

	// cudaMemcpy - results
	hipMemcpy(h_B, d_B, mem_size_B, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//////////////////////////////

	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start_G, stop_G);

	printf("Time took to compute matrix A of dimensions %d x %d  on GPU is %f ms \n \n \n", WA, HA, miliseconds);

	printf("=======input=========\n");
	for (int i = 0;i < HA;i++)
	{
		for (int j = 0;j < WA;j++)
		{
			printf("%f ", h_A[i*HA + j]);
		}
		printf("\n");
	}
	printf("\n\n=======kernel=========\n");
	for (int i = 0;i < HC;i++)
	{
		for (int j = 0;j < WC;j++)
		{
			printf("%f ", h_C[i*HC + j]);
		}
		printf("\n");
	}
	printf("\n\n=======GPU results=========\n");
	for (int i = 0;i < HB;i++)
	{
		for (int j = 0;j < WB;j++)
		{
			printf("%f ", h_B[i*HB + j]);
		}
		printf("\n");
	}

	hipEventRecord(start_C);
	// CPU Convolution function call
	Convolution_CPU(h_A, h_B, h_C, HA, WA, HB, WB, HC, WC);
	//////////////////////////////

	hipEventRecord(stop_C);
	hipEventSynchronize(stop_C);

	hipEventElapsedTime(&miliseconds, start_C, stop_C);
	printf("Time took to compute matrix A of dimensions %d x %d  on CPU is %f ms \n \n \n", WA, HA, miliseconds);

	printf("\n\n=======CPU results=========\n");
	for (int i = 0;i < HB;i++)
	{
		for (int j = 0;j < WB;j++)
		{
			printf("%f ", h_B[i*HB + j]);
		}
		printf("\n");
	}

	// memory release
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//////////////////////////////

	return EXIT_SUCCESS;
}
